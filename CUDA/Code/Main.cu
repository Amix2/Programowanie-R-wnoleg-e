#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include <cmath>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include "gputimer.h"
#include "cudaHeader.cuh"

int main(int argc, char* argv[])
{
	int mode = 4;
	if (argc >= 2)
		mode = atoi(argv[1]);
	switch (mode)
	{
	case 1:
		mainMatTranspose_1(argc-1, argv+1);
		break;
	case 2:
		mainImageScailing(argc-1, argv+1);
		break;
	case 3:
		mainReductionGlobal(argc - 1, argv + 1);
		break;
	case 4:
		mainWarmDivergence(argc - 1, argv + 1);
		break;
	}
	//mainVectorAdd(argc, argv);

	return 0;
}